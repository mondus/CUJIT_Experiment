#include "hip/hip_runtime.h"
﻿#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define NUM_THREADS 128
#define NUM_BLOCKS 32
#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      exit(1);                                                    \
    }                                                             \
  } while(0)
#define CUDA_SAFE_CALL(x)                                         \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      std::cerr << "\nerror: " #x " failed with error "           \
                << msg << '\n';                                   \
      exit(1);                                                    \
    }                                                             \
  } while(0)

const char* saxpy = R"###(    

#define NUM_THREADS 128
#define NUM_BLOCKS 32
#define N NUM_THREADS*NUM_BLOCKS

__device__ float dX[N];
__device__ float dY[N];
                                   
extern "C" __global__ void saxpy(float a, float *out, size_t n)   
{                                                               
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    out[tid] = a * dX[tid] + dY[tid];
  } 
}
)###";

int main()
{
	// Create an instance of hiprtcProgram with the SAXPY code string.
	hiprtcProgram prog;
	NVRTC_SAFE_CALL(
		hiprtcCreateProgram(&prog,         // prog
			saxpy,         // buffer
			"saxpy.cu",    // name
			0,             // numHeaders
			NULL,          // headers
			NULL));        // includeNames
	
	// register the named expressions
	NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "&dX"));
	//NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "&dY"));
						   
	// Compile the program with fmad disabled.
	// Note: Can specify GPU target architecture explicitly with '-arch' flag.
	
	hiprtcResult compileResult = hiprtcCompileProgram(prog,  // prog
		0,     // numOptions
		0); // options
	// Obtain compilation log from the program.
	size_t logSize;
	NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
	char* log = new char[logSize];
	NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log));
	std::cout << log << '\n';
	delete[] log;
	if (compileResult != HIPRTC_SUCCESS) {
		exit(1);
	}
	// Obtain PTX from the program.
	size_t ptxSize;
	NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
	char* ptx = new char[ptxSize];
	NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));
	printf("%s\n", ptx);
	// Load the generated PTX and get a handle to the SAXPY kernel.
	hipDevice_t cuDevice;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t kernel;
	CUDA_SAFE_CALL(hipInit(0));
	CUDA_SAFE_CALL(hipDeviceGet(&cuDevice, 0));
	CUDA_SAFE_CALL(hipCtxCreate(&context, 0, cuDevice));

	// Generate input for execution, and create output buffers.
	size_t n = NUM_THREADS * NUM_BLOCKS;
	size_t bufferSize = n * sizeof(float);
	float a = 5.1f;
	float* hX = new float[n];
	float* hX10 = new float[n];	// same as hX but x10
	float* hY = new float[n];
	float *hOut = new float[n];
	for (size_t i = 0; i < n; ++i) {
		hX[i] = static_cast<float>(i);
		hX10[i] = static_cast<float>(i*10);
		hY[i] = static_cast<float>(i * 2);
	}
	//alocated memory
	hipDeviceptr_t dX10, dOut;
	CUDA_SAFE_CALL(hipMalloc(&dOut, bufferSize));
	CUDA_SAFE_CALL(hipMalloc(&dX10, bufferSize));

	//options
	unsigned int SYMBOL_COUNT = 1;
	hipJitOption opts[3] = { hipJitOptionGlobalSymbolCount, hipJitOptionGlobalSymbolNames, hipJitOptionGlobalSymbolAddresses };
	const char* SYMBOL_NAMES[1] = { "dX" };
	void* SYMBOL_ADDRESSES[1] = { (void*)dX10 };
	void* optvals[3] = { &SYMBOL_COUNT, SYMBOL_NAMES, SYMBOL_ADDRESSES };

	// create and link module
	CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 3, opts, optvals));
	CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, "saxpy"));
	// device symbols
	hipDeviceptr_t dX, dY;
	// get mangled device name
	const char* dX_mangled_name;
	const char* dY_mangled_name;
	//NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "&dX", &dX_mangled_name));
	NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "&dY", &dY_mangled_name));
	// get device pointers
	//CUDA_SAFE_CALL(hipModuleGetGlobal(&dX, NULL, module, dX_mangled_name));
	CUDA_SAFE_CALL(hipModuleGetGlobal(&dY, NULL, module, dY_mangled_name));

	//CUDA_SAFE_CALL(hipMemcpyHtoD(dX, hX, bufferSize));
	CUDA_SAFE_CALL(hipMemcpyHtoD(dX10, hX10, bufferSize));
	CUDA_SAFE_CALL(hipMemcpyHtoD(dY, hY, bufferSize));
	// Destroy the program.
	NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
	// Execute SAXPY.
	void* args[] = { &a, &dOut, &n };
	CUDA_SAFE_CALL(
		hipModuleLaunchKernel(kernel,
			NUM_BLOCKS, 1, 1,    // grid dim
			NUM_THREADS, 1, 1,   // block dim
			0, NULL,             // shared mem and stream
			args, 0));           // arguments
	CUDA_SAFE_CALL(hipCtxSynchronize());
	// Retrieve and print output.
	CUDA_SAFE_CALL(hipMemcpyDtoH(hOut, dOut, bufferSize));
	for (size_t i = 0; i < n; ++i) {
		std::cout << a << " * " << hX[i] << " + " << hY[i]
			<< " = " << hOut[i] << '\n';
	}
	// Release resources.
	CUDA_SAFE_CALL(hipFree(dOut));
	CUDA_SAFE_CALL(hipModuleUnload(module));
	CUDA_SAFE_CALL(hipCtxDestroy(context));
	delete[] hX;
	delete[] hY;
	delete[] hOut;
	return 0;
}